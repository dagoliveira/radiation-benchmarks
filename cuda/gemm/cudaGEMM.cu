#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>

//#include "./log_helper.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"
//#include "cublas_v2.h"

#define MATRIX_PATH "/home/carol/TestGPU/GenerateGoldMatrix/Double_"

#define BLOCK_SIZE 32

#ifndef ITERACTIONS
#define ITERACTIONS 100000000000000000
#endif

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

int k=0; // N will be received on runtime

using namespace std;

string gold_matrix_path, a_matrix_path, b_matrix_path;

double *A;
double *B;
double *d_A;
double *d_B;
double *d_C;

   int lda, ldb, ldc;

double *GOLD;


FILE* f_A;
FILE* f_B;
FILE* f_GOLD;

FILE* file;
FILE* log_file;
FILE* timefile;

void UpdateTimestamp(){
	time_t timestamp = time(NULL);
	char time_s[50];
	sprintf(time_s, "%d", int(timestamp));

	char string[100] = "echo ";
	strcat(string, time_s);
	strcat(string, " > /home/carol/TestGPU/timestamp.txt");
	system(string);

//	printf("\n%s\n", string);
}


void log_error_detail(char* err)
{printf(err);
return;}

void end_log_file(){return;}

void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);
    
    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void ReadMatrixFromFile(){	
	
	int i;
	int j;
	double temp=0;
	double time = mysecond();
printf("open...");
	f_A = fopen(a_matrix_path.c_str(),"rb");
	f_B = fopen(b_matrix_path.c_str(),"rb");
	f_GOLD = fopen(gold_matrix_path.c_str(),"rb");
printf("read...");
	for(i=0; i<k; i++)
	{
		fread (&A[ lda * i ], sizeof(double)*k, 1, f_A);
		fread (&B[ lda * i ], sizeof(double)*k, 1, f_B);
		fread (&GOLD[ lda * i ], sizeof(double)*k, 1, f_GOLD);
		//for(j=0; j<n; j++){
//
//			A[i + lda * j] = 0.0;
//			B[j + ldb * i] = 0.0;
//
//			GOLD[i + ldc * j] = 0.0; 
//
//			fread(&A[i + lda * j],sizeof(double), 1, f_A);
//			fread(&B[j + ldb * i],sizeof(double), 1, f_B);
//			fread(&GOLD[i + ldc * j],sizeof(double), 1, f_GOLD);
//			
//		}
	}	printf("\n");
	for (i=0; i<k; i++)
	{ 
		for (j=0; (j<k)&&(j<i); j++)
		{
			temp = GOLD [i + ldc * j];
			GOLD [i + ldc * j] = GOLD [j + ldc * i];
			GOLD [j + ldc * i] = temp;
		}
	}
printf("ok in %f\n", mysecond() - time);

//A[45] = 5.5;
	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);
}

__device__ int kerrors;

__global__ void GoldChkKernel (double *gk, double *ck, int n)//, int *kerrors)
{
	//ck[4] = 4.5;
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                                      
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y; 
	if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
//	if (gk[ty*n + tx]!=ck[ty*n + tx])
		atomicAdd(&kerrors, 1);//kerrors++;

}



int main( int argc, char* argv[] )
{

	
	hipError_t malloc_mem1;
	hipError_t malloc_a;
	const char *erro_malloc;

	int ea=0; //wrong integers in the current loop
	int t_ea=0; //total number of wrong integers

	double total_time = 0.0;

	const double alpha = 1.0;
	const double beta = 1.0;

	char transa = 't', transb = 't';
	int i, j, loop2;

	int kernel_errors=0;
	int zero = 0;


	int sizea, sizeb, sizec;

	////////////////////////////////////////////////////
	////////////////////GET PARAM///////////////////////
	if (argc!=2) {
		printf ("Enter the required input. (1024/2048/4096/8192)\n");
		exit (-1);
	}
	k = atoi (argv[1]);
	if (((k%32)!=0)||(k<0)){
		printf ("Enter a valid input. (k=%i)\n", k);
		exit (-1);
	}
	string matrix_size_str(argv[1]);

	a_matrix_path = MATRIX_PATH;
	b_matrix_path = MATRIX_PATH;
	gold_matrix_path = MATRIX_PATH;
	a_matrix_path += "A_8192.matrix";
	b_matrix_path += "B_8192.matrix";
	gold_matrix_path += "GOLD_" + matrix_size_str + ".matrix";

	//////////BLOCK and GRID size///////////////////////
	int gridsize = k/BLOCK_SIZE < 1 ? 1 : k/BLOCK_SIZE;
	int blocksize = k/BLOCK_SIZE < 1 ? k : BLOCK_SIZE;
	dim3 dimBlock(blocksize,blocksize);
	dim3 dimGrid(gridsize,gridsize);
	////////////////////////////////////////////////////


	char test_info[90];
	snprintf(test_info, 90, "size:%d", k);
	////start_log_file("cudaGEMM", test_info);


	lda = max( 1, k + 16 );
	sizea = lda * k;
	ldb = max( 1, k + 16 );
	sizeb = ldb * k;
	ldc = max( 1, k + 16 );
	sizec = ldc * k;

	A = ( double* ) malloc( sizea * sizeof( double ) );
	B = ( double* ) malloc( sizeb * sizeof( double ) );

	GOLD = ( double* ) malloc( sizec * sizeof( double ) );

	kernel_errors=0;
	
	GetDevice();
	
	ReadMatrixFromFile();

	//A[72] = 7.2;

	printf( "cublasDGEMM\n" );

   
	for(loop2=0; loop2<ITERACTIONS; loop2++)
	{


		malloc_a = hipMalloc( ( void** ) &d_A, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error a"); end_log_file(); return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_B, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error b"); end_log_file(); return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_C, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error c"); end_log_file(); return 1;} //mem allocate failure


		malloc_mem1 = hipMemcpy( d_C, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // ZERA C
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error mem load c "); end_log_file(); return 1;}
	
		malloc_mem1 = hipMemcpy( d_A, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error mem load a "); end_log_file(); return 1;}

		malloc_mem1 = hipMemcpy( d_B, B, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {log_error_detail("error mem load b "); end_log_file(); return 1;}

		kernel_errors=0;
		//cublasHandle_t blashandle;
		//cublasCreate(&blashandle);
	
		printf("cublasDgemm... k=%d transa=%c transb=%c lda=%d ldb=%d ldc=%d\n", k, transa, transb, lda, ldb, ldc);
		////start_iteration();
		//cublasDgemm( blashandle, (cublasOperation_t)transa, (cublasOperation_t)transb,
		cublasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
			   k, k, k,
			   alpha,
			   d_A, lda,
			   d_B, ldb,
			   beta,
			   d_C, ldc );
		printf("\nend\n");
		hipDeviceSynchronize();
		////end_iteration();

		malloc_mem1 = hipMemcpy(d_A, GOLD, sizea * sizeof( double ), hipMemcpyHostToDevice );
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error mem load a %s", erro_malloc); fprintf(file, "error mem load a %s", erro_malloc); return 1;}

		hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));

		GoldChkKernel<<<dimGrid,dimBlock>>>(d_A, d_C, ldc);


		hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors), sizeof(unsigned int));
	
	 
		/////////////UPDATE TIMESTAMP///////////////////
	//	UpdateTimestamp();
		////////////////////////////////////////////////
		
		////log_error_count(kernel_errors);

		if (kernel_errors!=0)
		{

			printf("\n kernel error: %d\n", kernel_errors);

			malloc_mem1 = hipMemcpy(A, d_C, sizec * sizeof( double ), hipMemcpyDeviceToHost);
			erro_malloc = hipGetErrorString(malloc_mem1);
			if(strcmp(erro_malloc, "no error") != 0)
				{printf("error mem load a %s", erro_malloc); fprintf(file, "error mem load a %s", erro_malloc); return 1;}
			char error_detail[150];

			for(i=0; (i<k) && (ea < 500); i++)
			{
				for(j=0; (j<k) && (ea < 500); j++)
				{
					if ((fabs((A[i+ldc*j]-GOLD[i+ldc*j])/A[i+ldc*j]) > 0.0000000001)||(fabs((A[i+ldc*j]-GOLD[i+ldc*j])/GOLD[i+ldc*j]) > 0.0000000001))
					{
						snprintf(error_detail, 150, "p: [%d, %d], r: %1.16e, e: %1.16e", i, j, A[i + ldc * j], GOLD[i + ldc * j]);
						////log_error_detail(error_detail);
						//ea++;			
						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + ldc * j], GOLD[i + ldc * j], t_ea);
										
					}
				}
			}

				ReadMatrixFromFile();	
		}



		if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("\ntest number: %d", loop2);
		}
		else
		{
			printf(".");
		}



		hipFree( d_A );
		hipFree( d_B );
		hipFree( d_C );
	}

	free( A );
	free( B );

	end_log_file();

	return 0;
}
